#include "hip/hip_runtime.h"
#include <cmath>
#include "def.h"
#include "../../include/smpl/smpl.h"

namespace smpl {
    namespace device {
        __global__ void PoseBlend1(float *theta, int jointnum,
                                   float *poseRotation, float *restPoseRotation) {
            int i = blockIdx.x;
            int j = threadIdx.x;

            int ind = i * jointnum * 3 + j * 3;
            float norm = std::sqrt(theta[ind] * theta[ind] + theta[ind + 1] * theta[ind + 1] + theta[ind + 2] * theta[ind + 2]);
            float sin = std::sin(norm);
            float cos = std::cos(norm);
            theta[ind] /= norm;
            theta[ind + 1] /= norm;
            theta[ind + 2] /= norm; // axes

            float skew[9];
            skew[0] = 0;
            skew[1] = -1 * theta[ind + 2];
            skew[2] = theta[ind + 1];
            skew[3] = theta[ind + 2];
            skew[4] = 0;
            skew[5] = -1 * theta[ind];
            skew[6] = -1 * theta[ind + 1];
            skew[7] = theta[ind];
            skew[8] = 0;

            ind = ind * 3;
            for (int p = 0; p < 0; p++)
                poseRotation[ind + p] = 0;
            poseRotation[ind] = 1;
            poseRotation[ind + 4] = 1;
            poseRotation[ind + 8] = 1;
            for (int k1 = 0; k1 < 3; k1++)
                for (int k2 = 0; k2 < 3; k2++) {
                    int k = k1 * 3 + k2;
                    poseRotation[ind + k] += skew[k] * sin;
                    float num = 0;
                    for (int l = 0; l < 3; l++)
                        num += skew[k1 * 3 + l] * skew[l * 3 + k2];
                    poseRotation[ind + k] += (1 - cos) * num;// (N, 24, 3, 3)
                }

            for (int k = 0; k < 9; k++)
                restPoseRotation[ind + k] = 0;
            restPoseRotation[ind] = 1;
            restPoseRotation[ind + 4] = 1;
            restPoseRotation[ind + 8] = 1;
        }

        __global__ void PoseBlend2(float *poseRotation, float *poseBlendBasis, float *restPoseRotation, int jointnum, int vertexnum,
                                   float *poseBlendShape) {
            int i = blockIdx.x; // batch size
            int j = threadIdx.x; // vertex num
            for (int k = 0; k < 3; k++) {
                poseBlendShape[i * vertexnum * 3 + j * 3 + k] = 0;
                for (int l = 0; l < 207; l++)
                    poseBlendShape[i * vertexnum * 3 + j * 3 + k] +=
                            (poseRotation[i * jointnum * 9 + l + 9] - restPoseRotation[i * jointnum * 9 + l + 9])
                                                * poseBlendBasis[j * 3 * 207 + k * 207 + l];
            }
        }

        __global__ void ShapeBlend(float *beta, float *shapeBlendBasis, int vertexnum, int shapebasisdim,
                                   float *shapeBlendShape) {
            int i = blockIdx.x;
            int j = threadIdx.x;
            for (int k = 0; k < 3; k++) {
                shapeBlendShape[i * vertexnum * 3 + j * 3 + k] = 0;
                for (int l = 0; l < shapebasisdim; l++)
                    shapeBlendShape[i * vertexnum * 3 + j * 3 + k] += beta[i * shapebasisdim + l] *
                            shapeBlendBasis[j * shapebasisdim * 3 + k * shapebasisdim + l];// (N, 6890, 3)
            }
        }

        __global__ void RegressJoints1(float *templateRestShape, float *shapeBlendShape, float *poseBlendShape,int vertexnum,
                                       float *restShape) {
            int i = blockIdx.x;
            int j = threadIdx.x;
            for (int k = 0; k < 3; k++) {
                int ind = i * vertexnum * 3 + j * 3 + k;
                restShape[ind] = templateRestShape[ind] + shapeBlendShape[ind] + poseBlendShape[ind];
            }
        }

        __global__ void RegressJoints2(float *templateRestShape, float *shapeBlendShape, float *jointRegressor, int jointnum, int vertexnum,
                                       float *joints) {
            int i = blockIdx.x;
            int j = threadIdx.x;
            for (int l = 0; l < 3; l++) {
                joints[i * jointnum * 3 + j * 3 + l] = 0;
                for (int k = 0; k < vertexnum; k++)
                    joints[i * jointnum * 3 + j * 3 + l] += (templateRestShape[i * vertexnum * 3 + k * 3 + l] +
                                                             shapeBlendShape[i * vertexnum * 3 + k * 3 + l]) *
                                                            jointRegressor[j * vertexnum + k];
            }
        }

        __global__ void LocalTransform(float *joints, int64_t *kinematicTree, float *poseRotation, int jointnum,
                                       float *localTransformations) {
            // joints [batchsize][jointnum][3]
            // poseRotHomo [batchsize][JOINTS_NUM][4][3]
            // kinematicTree [2][jointnum]
            int j = blockIdx.x;
            int i = threadIdx.x;
            //copy data from poseRotation
            for (int k = 0; k < 3; k++)
                for (int l = 0; l < 3; l++)
                    localTransformations[j * jointnum * 16 + i * 16 + k * 4 + l] =
                            poseRotation[j * jointnum * 9 + i * 9 + k * 3 + l];
            for (int l = 0; l < 3; l++)
                localTransformations[j * jointnum * 16 + i * 16 + 3 * 4 + l] = 0;
            // data from joints
            int ancestor = kinematicTree[i];
            for (int k = 0; k < 3; k++)
                localTransformations[j * jointnum * 16 + i * 16 + k * 4 + 3] =
                        i != 0 ? joints[j * jointnum * 3 + i * 3 + k] - joints[j * jointnum * 3 + ancestor * 3 + k]
                        : joints[j * jointnum * 3 + k];
            localTransformations[j * jointnum * 16 + i * 16 + 3 * 4 + 3] = 1;
        }


        __global__ void GlobalTransform(float *localTransformations, int64_t *kinematicTree, int jointnum, int batchsize,
                                        float *globalTransformations) {
            //global transformations [N][24][4][4]
            for (int i = 0; i < batchsize; i++)
                for (int k = 0; k < 4; k++)
                    for (int l = 0; l < 4; l++)
                        globalTransformations[i * jointnum * 16 + k * 4 + l] = localTransformations[i * jointnum * 16 + k * 4 + l];
            for (int j = 1; j < jointnum; j++) {
                int anc = kinematicTree[j];
                for (int i = 0; i < batchsize; i++)
                    for (int k = 0; k < 4; k++)
                        for (int l = 0; l < 4; l++) {
                            globalTransformations[i * jointnum * 16 + j * 16 + k * 4 + l] = 0;
                            for (int t = 0; t < 4; t++)
                                globalTransformations[i * jointnum * 16 + j * 16 + k * 4 + l] +=
                                        globalTransformations[i * jointnum * 16 + anc * 16 + k * 4 + t] *
                                        localTransformations[i * jointnum * 16 + j * 16 + t * 4 + l];
                        }
            }
        }

        __global__ void Transform(float *globalTransformations, float *joints, int jointnum) {
            int i = blockIdx.x;
            int j = threadIdx.x;

            float elim[3];
            for (int k = 0; k < 3; k++)
                elim[k] = joints[i * jointnum * 3 + j * 3 + k];

//            float elim[3];
//            for (int k = 0; k < 3; k++) {
//                elim[k] = 0;
//                for (int t = 0; t < 3; t++)
//                    elim[k] += globalTransformations[i * jointnum * 16 + j * 16 + t * 4 + k * 4] *
//                               joints[i * jointnum * 3 + j * 3 + t];
//            }
            for (int k = 0; k < 3; k++)
                globalTransformations[i * jointnum * 16 + j * 16 + k * 4 + 3] -= elim[k];
        }

        __global__ void Skinning(float *restShape, float *transformation, float *weights, int batchsize, int vertexnum, int jointnum,
                                 float *vertices) {
            //restShape [batchsize][vertexnum][3]
            //transformation [batchsize][jointnum][4][4]
            //weights [vertexnum][jointnum]

            // linear blend skinning
            for (int i = 0; i < batchsize; i++)
                for (int j = 0; j < vertexnum; j++) {
                    float coeffs[16] = {0};
                    for (int k = 0; k < 4; k++)
                        for (int l = 0; l < 4; l++)
                            for (int t = 0; t < jointnum; t++)
                                coeffs[k * 4 + l] +=weights[j * jointnum + t] * transformation[i * jointnum * 16 + t * 16 + k * 4 + l];

                    float homoW = coeffs[15];
                    for (int t = 0; t < 3; t++)
                        homoW += coeffs[12 + t] * restShape[i * vertexnum * 3 + j * 3 + t];
                    for (int k = 0; k < 3; k++) {
                        vertices[i * vertexnum * 3 + j * 3 + k] = coeffs[k * 4 + 3];
                        for (int t = 0; t < 3; t++)
                            vertices[i * vertexnum * 3 + j * 3 + k] += coeffs[k * 4 + t] * restShape[i * vertexnum * 3 + j * 3 + t];
                        vertices[i * vertexnum * 3 + j * 3 + k] /= homoW;
                    }
                }
        }
    }

    void SMPL::loadToDevice() {
        ///BLEND SHAPE
        hipMalloc((void **) &d_poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_poseBlendBasis, m__poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void **) &d_shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_shapeBlendBasis, m__shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        ///REGRESS JOINTS
        hipMalloc((void **) &d_templateRestShape, VERTEX_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float));
        hipMemcpy(d_templateRestShape, m__templateRestShape, VERTEX_NUM * 3  * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_jointRegressor, m__jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float), hipMemcpyHostToDevice);
        ///WORLD TRANSFORMATIONS
        hipMalloc((void **) &d_kinematicTree, 2 * JOINT_NUM * sizeof(int64_t));
        hipMemcpy(d_kinematicTree, m__kinematicTree, 2 * JOINT_NUM * sizeof(int), hipMemcpyHostToDevice);
        ///SKINNING
        hipMalloc((void **) &d_weights, VERTEX_NUM * JOINT_NUM * sizeof(float));
        hipMemcpy(d_weights, m__weights, VERTEX_NUM * JOINT_NUM * sizeof(float), hipMemcpyHostToDevice);
    }

    void SMPL::run(float *beta, float *theta) {
        auto bs = blendShape(beta, theta);
        auto d_poseRotation = std::get<0>(bs);
        auto d_restPoseRotation = std::get<1>(bs);
        auto d_poseBlendShape = std::get<2>(bs);
        auto d_shapeBlendShape = std::get<3>(bs);

        auto rj = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        auto d_restShape = std::get<0>(rj);
        auto d_joints = std::get<1>(rj);
        //auto [d_poseRotation, d_restPoseRotation, d_poseBlendShape, d_shapeBlendShape] = blendShape(beta, theta);
        //auto [d_restShape, d_joints] = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        hipFree(d_shapeBlendShape);
        hipFree(d_poseBlendShape);
        auto d_transformation = transform(d_poseRotation, d_joints);
        hipFree(d_poseRotation);
        hipFree(d_joints);
        skinning(d_restShape, d_transformation);
        hipFree(d_restShape);
        hipFree(d_transformation);
    }

    std::tuple<float *, float *, float *, float *> SMPL::blendShape(float *theta, float *beta) {
        ///BLEND SHAPE
        float *d_theta, *d_poseRotation, *d_restPoseRotation, *d_poseBlendShape;
        hipMalloc((void **) &d_theta, BATCH_SIZE * JOINT_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_poseRotation, BATCH_SIZE * JOINT_NUM * 9 * sizeof(float));
        hipMalloc((void **) &d_restPoseRotation, BATCH_SIZE * JOINT_NUM * 9 * sizeof(float));
        hipMalloc((void **) &d_poseBlendShape, BATCH_SIZE * VERTEX_NUM * 3 * sizeof(float));
        hipMemcpy(d_theta, theta, BATCH_SIZE * JOINT_NUM * 3 * sizeof(float), hipMemcpyHostToDevice);

        device::PoseBlend1<<<BATCH_SIZE,JOINT_NUM>>>(d_theta, JOINT_NUM, d_poseRotation, d_restPoseRotation);
        device::PoseBlend2<<<BATCH_SIZE,VERTEX_NUM>>>(d_poseRotation, d_poseBlendBasis, d_restPoseRotation,
                                                        JOINT_NUM, VERTEX_NUM, d_poseBlendShape);

        float *d_beta, *d_shapeBlendShape;
        hipMalloc((void **) &d_beta, BATCH_SIZE * SHAPE_BASIS_DIM * sizeof(float));
        hipMalloc((void **) &d_shapeBlendShape, BATCH_SIZE * VERTEX_NUM * 3 * sizeof(float));
        hipMemcpy(d_beta, beta, BATCH_SIZE * SHAPE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);

        device::ShapeBlend<<<BATCH_SIZE,VERTEX_NUM>>>(d_beta, d_shapeBlendBasis, VERTEX_NUM, SHAPE_BASIS_DIM, d_shapeBlendShape);

        hipFree(d_theta);
        hipFree(d_beta);

        return {d_poseRotation, d_restPoseRotation, d_poseBlendShape, d_shapeBlendShape};
    }

    std::tuple<float *, float *> SMPL::regressJoints(float *d_shapeBlendShape, float *d_poseBlendShape) {
        ///REGRESS JOINTS
        float *d_joints, *d_restShape;
        hipMalloc((void **) &d_joints, BATCH_SIZE * JOINT_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_restShape, BATCH_SIZE * VERTEX_NUM * 3 * sizeof(float));

        device::RegressJoints1<<<BATCH_SIZE,VERTEX_NUM>>>(d_templateRestShape, d_shapeBlendShape, d_poseBlendShape,
                VERTEX_NUM, d_restShape);
        device::RegressJoints2<<<BATCH_SIZE,JOINT_NUM>>>(d_templateRestShape, d_shapeBlendShape, d_jointRegressor,
                JOINT_NUM, VERTEX_NUM, d_joints);

        return {d_restShape, d_joints};
    }

    float *SMPL::transform(float *d_poseRotation, float *d_joints) {
        ///WORLD TRANSFORMATIONS
        float *d_localTransformations, *d_globalTransformations;
        hipMalloc((void **) &d_localTransformations, BATCH_SIZE * JOINT_NUM * 16 * sizeof(float));
        hipMalloc((void **) &d_globalTransformations, BATCH_SIZE * JOINT_NUM * 16 * sizeof(float));

        device::LocalTransform<<<BATCH_SIZE,JOINT_NUM>>>(d_joints, d_kinematicTree, d_poseRotation, JOINT_NUM, d_localTransformations);
        device::GlobalTransform<<<1,1>>>(d_localTransformations, d_kinematicTree, JOINT_NUM, BATCH_SIZE, d_globalTransformations);
        device::Transform<<<BATCH_SIZE,JOINT_NUM>>>(d_globalTransformations, d_joints, JOINT_NUM);

        hipFree(d_localTransformations);
        return d_globalTransformations;
    }

    void SMPL::skinning(float *d_restShape, float *d_transformation) {
        ///SKINNING
        float *d_vertices;
        hipMalloc((void **) &d_vertices, BATCH_SIZE * VERTEX_NUM * 3 * sizeof(float));

        device::Skinning<<<BATCH_SIZE,VERTEX_NUM>>>(d_restShape, d_transformation, d_weights,
                BATCH_SIZE, VERTEX_NUM, JOINT_NUM, d_vertices);

        hipMemcpy(m__result_vertices, d_vertices, BATCH_SIZE * VERTEX_NUM * 3 * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_vertices);
    }

    SMPL::~SMPL() {
        //CPU
//        free(m__faceIndices);
//        free(m__shapeBlendBasis);
//        free(m__poseBlendBasis);
//        free(m__templateRestShape);
//        free(m__jointRegressor);
//        free(m__kinematicTree);
//        free(m__weights);
//        free(m__result_vertices);

        ///GPU
//        hipFree(d_poseBlendBasis);
//        hipFree(d_shapeBlendBasis);
//        hipFree(d_templateRestShape);
//        hipFree(d_jointRegressor);
//        hipFree(d_weights);
//        hipFree(d_kinematicTree);
    }
}