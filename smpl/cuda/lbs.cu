#include "hip/hip_runtime.h"
#include <cmath>
#include "../def.h"
#include "../smpl.h"

namespace smpl {
    namespace device {
        __global__ void FindKNN1(float *templateRestShape, float *shapeBlendShape, int vertexnum, float *curvertices,
                                 float *dist) {
            //int i = blockIdx.x;
            int i = 0;
            int j = threadIdx.x;
            int ind = i * vertexnum + j;
            dist[ind] = 0;
            for (int k = 0; k < 3; k++) {
                float restShape = templateRestShape[j * 3 + k] + shapeBlendShape[j * 3 + k];
                dist[ind] += (curvertices[i * 3 + k] - restShape) * (curvertices[i * 3 + k] - restShape);
            }
        }

        __global__ void FindKNN2(float *dist, int vertexnum,
                                 int *ind) {
            //int i = threadIdx.x;

            //sort them
            ind[0] = 0;
            ind[1] = 1;
            ind[2] = 2;
            ind[3] = 3;

            for (int l = 0; l < 4; l++)
                for (int p = 0; p < 3 - l; p++)
                    if (dist[ind[p]] > dist[ind[p + 1]]) {
                        int tmp = ind[p];
                        ind[p] = ind[p + 1];
                        ind[p + 1] = tmp;
                    }

            //find first 4 minimum distances
            for (int k = 4; k < vertexnum; k++)
                for (int t = 0; t < 4; t++)
                    if (dist[k] < dist[ind[t]]) {
                        for (int l = 3; l > t; l--)
                            ind[l] = ind[l - 1];
                        ind[t] = k;
                        continue;
                    }
        }

        __global__ void CalculateWeights(float *dist, float *weights, int *ind, int jointnum, int vertexnum,
                                         float *new_weights) {
            int j = threadIdx.x; // num of weight
            //int i = blockIdx.x;
            int i = 0; // num of vertex

            new_weights[i * jointnum + j] = 0;
            float weight = 0;
            for (int k = 0; k < 4; k++) {
                weight += dist[ind[i * 4 + k]];
                new_weights[i * jointnum + j] += dist[ind[i * 4 + k]] *
                        weights[ind[i * 4 + k] * jointnum + j];
            }
            new_weights[i * jointnum + j] /= weight;
        }
    }

    // linear blend skinning for vertex [3]
    float *SMPL::lbs_for_custom_vertices(float *beta, float *theta, float *vertex) {
        auto d_shapeBlendShape = shapeBlendShape(beta);

        float *d_dist;
        hipMalloc((void **) &d_dist, VERTEX_NUM * sizeof(float));
        int *d_ind;
        hipMalloc((void **) &d_ind, 4 * sizeof(int));
        float *d_vertex;
        hipMalloc((void **) &d_vertex, 3 * sizeof(float));
        hipMemcpy(d_vertex, vertex, 3 * sizeof(float), hipMemcpyHostToDevice);
        float *d_cur_weights;
        hipMalloc((void **) &d_cur_weights, JOINT_NUM * sizeof(float));

        // find k nearest neigbours
        device::FindKNN1<<<1,VERTEX_NUM>>>(d_templateRestShape, d_shapeBlendShape, VERTEX_NUM, d_vertex, d_dist);
        device::FindKNN2<<<1,1>>>(d_dist, VERTEX_NUM, d_ind);
        //now we can calculate weights
        device::CalculateWeights<<<1,JOINT_NUM>>>(d_dist, m__weights, d_ind,  JOINT_NUM, VERTEX_NUM, d_cur_weights);
        hipFree(d_shapeBlendShape);
        hipFree(d_dist);
        hipFree(d_ind);

        auto res = run(beta, theta, d_cur_weights, d_vertex, 1);
        hipFree(d_cur_weights);
        hipFree(d_vertex);

        return res;
    }
}