#include <cmath>
#include "../def.h"
#include "../smpl.h"

namespace smpl {
    void SMPL::loadToDevice() {
        ///BLEND SHAPE
        hipMalloc((void **) &d_poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_poseBlendBasis, m__poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void **) &d_shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_shapeBlendBasis, m__shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        ///REGRESS JOINTS
        hipMalloc((void **) &d_templateRestShape, VERTEX_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float));
        hipMemcpy(d_templateRestShape, m__templateRestShape, VERTEX_NUM * 3  * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_jointRegressor, m__jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float), hipMemcpyHostToDevice);
        ///WORLD TRANSFORMATIONS
        hipMalloc((void **) &d_kinematicTree, 2 * JOINT_NUM * sizeof(int64_t));
        hipMemcpy(d_kinematicTree, m__kinematicTree, 2 * JOINT_NUM * sizeof(int), hipMemcpyHostToDevice);
        ///SKINNING
        hipMalloc((void **) &d_weights, VERTEX_NUM * JOINT_NUM * sizeof(float));
        hipMemcpy(d_weights, m__weights, VERTEX_NUM * JOINT_NUM * sizeof(float), hipMemcpyHostToDevice);
    }

    float *SMPL::run(float *beta, float *theta, float *d_custom_weights, float *d_vertices = nullptr, float vertexnum = 0) {
        auto pbs = poseBlendShape(theta);
        auto d_poseRotation = std::get<0>(pbs);
        auto d_restPoseRotation = std::get<1>(pbs);
        auto d_poseBlendShape = std::get<2>(pbs);

        auto d_shapeBlendShape = shapeBlendShape(beta);

        auto rj = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        auto d_restShape = std::get<0>(rj);
        auto d_joints = std::get<1>(rj);
        hipFree(d_shapeBlendShape);
        hipFree(d_poseBlendShape);
        auto d_transformation = transform(d_poseRotation, d_joints);
        hipFree(d_poseRotation);
        hipFree(d_joints);

        if (d_vertices == nullptr) {
            d_vertices = d_restShape;
            vertexnum = VERTEX_NUM
        }

        float *res = skinning(d_transformation, d_custom_weights, d_vertices, vertexnum);
        hipFree(d_restShape);
        hipFree(d_transformation);

        return res;
    }

    float *SMPL::lbs_for_model(float *beta, float *theta) {
        return run(beta, theta, m__weights);
    }

    SMPL::~SMPL() {
        ///CPU
        if (m__faceIndices != nullptr)
            free(m__faceIndices);
        if (m__shapeBlendBasis != nullptr)
            free(m__shapeBlendBasis);
        if (m__poseBlendBasis != nullptr)
            free(m__poseBlendBasis);
        if (m__templateRestShape != nullptr)
            free(m__templateRestShape);
        if (m__jointRegressor != nullptr)
            free(m__jointRegressor);
        if (m__kinematicTree != nullptr)
            free(m__kinematicTree);
        if (m__weights != nullptr)
            free(m__weights);

        ///GPU
        if (d_poseBlendBasis != nullptr)
            hipFree(d_poseBlendBasis);
        if (d_shapeBlendBasis != nullptr)
            hipFree(d_shapeBlendBasis);
        if (d_templateRestShape != nullptr)
            hipFree(d_templateRestShape);
        if (d_jointRegressor != nullptr)
            hipFree(d_jointRegressor);
        if (d_weights != nullptr)
            hipFree(d_weights);
        if (d_kinematicTree != nullptr)
            hipFree(d_kinematicTree);
    }
}