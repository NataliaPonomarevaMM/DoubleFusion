#include <cmath>
#include "../def.h"
#include "../smpl.h"

namespace smpl {
    void SMPL::loadToDevice() {
        ///BLEND SHAPE
        hipMalloc((void **) &d_poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_poseBlendBasis, m__poseBlendBasis, VERTEX_NUM * 3 * POSE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void **) &d_shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float));
        hipMemcpy(d_shapeBlendBasis, m__shapeBlendBasis, VERTEX_NUM * 3 * SHAPE_BASIS_DIM * sizeof(float), hipMemcpyHostToDevice);
        ///REGRESS JOINTS
        hipMalloc((void **) &d_templateRestShape, VERTEX_NUM * 3 * sizeof(float));
        hipMalloc((void **) &d_jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float));
        hipMemcpy(d_templateRestShape, m__templateRestShape, VERTEX_NUM * 3  * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_jointRegressor, m__jointRegressor, JOINT_NUM * VERTEX_NUM * sizeof(float), hipMemcpyHostToDevice);
        ///WORLD TRANSFORMATIONS
        hipMalloc((void **) &d_kinematicTree, 2 * JOINT_NUM * sizeof(int64_t));
        hipMemcpy(d_kinematicTree, m__kinematicTree, 2 * JOINT_NUM * sizeof(int), hipMemcpyHostToDevice);
        ///SKINNING
        hipMalloc((void **) &d_weights, VERTEX_NUM * JOINT_NUM * sizeof(float));
        hipMemcpy(d_weights, m__weights, VERTEX_NUM * JOINT_NUM * sizeof(float), hipMemcpyHostToDevice);
    }

    float *SMPL::run(float *beta, float *theta) {
        auto bs = blendShape(beta, theta);
        auto d_poseRotation = std::get<0>(bs);
        auto d_restPoseRotation = std::get<1>(bs);
        auto d_poseBlendShape = std::get<2>(bs);
        auto d_shapeBlendShape = std::get<3>(bs);

        auto rj = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        auto d_restShape = std::get<0>(rj);
        auto d_joints = std::get<1>(rj);
        //auto [d_poseRotation, d_restPoseRotation, d_poseBlendShape, d_shapeBlendShape] = blendShape(beta, theta);
        //auto [d_restShape, d_joints] = regressJoints(d_shapeBlendShape, d_poseBlendShape);
        hipFree(d_shapeBlendShape);
        hipFree(d_poseBlendShape);
        auto d_transformation = transform(d_poseRotation, d_joints);
        hipFree(d_poseRotation);
        hipFree(d_joints);
        float *res = skinning(d_restShape, d_transformation);
        hipFree(d_restShape);
        hipFree(d_transformation);

        return res;
    }

    SMPL::~SMPL() {
        ///CPU
        if (m__faceIndices != nullptr)
            free(m__faceIndices);
        if (m__shapeBlendBasis != nullptr)
            free(m__shapeBlendBasis);
        if (m__poseBlendBasis != nullptr)
            free(m__poseBlendBasis);
        if (m__templateRestShape != nullptr)
            free(m__templateRestShape);
        if (m__jointRegressor != nullptr)
            free(m__jointRegressor);
        if (m__kinematicTree != nullptr)
            free(m__kinematicTree);
        if (m__weights != nullptr)
            free(m__weights);

        ///GPU
        if (d_poseBlendBasis != nullptr)
            hipFree(d_poseBlendBasis);
        if (d_shapeBlendBasis != nullptr)
            hipFree(d_shapeBlendBasis);
        if (d_templateRestShape != nullptr)
            hipFree(d_templateRestShape);
        if (d_jointRegressor != nullptr)
            hipFree(d_jointRegressor);
        if (d_weights != nullptr)
            hipFree(d_weights);
        if (d_kinematicTree != nullptr)
            hipFree(d_kinematicTree);
    }
}